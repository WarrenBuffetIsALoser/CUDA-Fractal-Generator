#include "hip/hip_runtime.h"
// CUDA
#include "hip/hip_runtime.h"
#include ""

// OPEN_GL
#include <OPEN_GL/glew.h>

// SDL2
#include <SDL2/SDL.h>

#include "SDL_Display.h"
#include "shader.h"
#include <stdio.h>

typedef struct {
	double r;       // a fraction between 0 and 1
	double g;       // a fraction between 0 and 1
	double b;       // a fraction between 0 and 1
} rgb;

typedef struct {
	double h;       // angle in degrees
	double s;       // a fraction between 0 and 1
	double v;       // a fraction between 0 and 1
} hsv;

static hsv   rgb2hsv(rgb in);
static rgb   hsv2rgb(hsv in);

hsv rgb2hsv(rgb in)
{
	hsv         out;
	double      min, max, delta;

	min = in.r < in.g ? in.r : in.g;
	min = min < in.b ? min : in.b;

	max = in.r > in.g ? in.r : in.g;
	max = max > in.b ? max : in.b;

	out.v = max;                                // v
	delta = max - min;
	if (delta < 0.00001)
	{
		out.s = 0;
		out.h = 0; // undefined, maybe nan?
		return out;
	}
	if (max > 0.0) { // NOTE: if Max is == 0, this divide would cause a crash
		out.s = (delta / max);                  // s
	}
	else {
		// if max is 0, then r = g = b = 0              
		// s = 0, h is undefined
		out.s = 0.0;
		out.h = NAN;                            // its now undefined
		return out;
	}
	if (in.r >= max)                           // > is bogus, just keeps compilor happy
		out.h = (in.g - in.b) / delta;        // between yellow & magenta
	else
		if (in.g >= max)
			out.h = 2.0 + (in.b - in.r) / delta;  // between cyan & yellow
		else
			out.h = 4.0 + (in.r - in.g) / delta;  // between magenta & cyan

	out.h *= 60.0;                              // degrees

	if (out.h < 0.0)
		out.h += 360.0;

	return out;
}

rgb hsv2rgb(hsv in)
{
	double      hh, p, q, t, ff;
	long        i;
	rgb         out;

	if (in.s <= 0.0) {       // < is bogus, just shuts up warnings
		out.r = in.v;
		out.g = in.v;
		out.b = in.v;
		return out;
	}
	hh = in.h;
	if (hh >= 360.0) hh = 0.0;
	hh /= 60.0;
	i = (long)hh;
	ff = hh - i;
	p = in.v * (1.0 - in.s);
	q = in.v * (1.0 - (in.s * ff));
	t = in.v * (1.0 - (in.s * (1.0 - ff)));

	switch (i) {
	case 0:
		out.r = in.v;
		out.g = t;
		out.b = p;
		break;
	case 1:
		out.r = q;
		out.g = in.v;
		out.b = p;
		break;
	case 2:
		out.r = p;
		out.g = in.v;
		out.b = t;
		break;

	case 3:
		out.r = p;
		out.g = q;
		out.b = in.v;
		break;
	case 4:
		out.r = t;
		out.g = p;
		out.b = in.v;
		break;
	case 5:
	default:
		out.r = in.v;
		out.g = p;
		out.b = q;
		break;
	}
	return out;
}

int WIDTH = 2000;
int HEIGHT = 2000;

double maxIterations = 100;

double min_valx = -2;
double max_valx = 0;

double min_valy = -1;
double max_valy = 1;

float *d_data;
float *texData;

int threadsPerBlock = 512;
int blocksPerGrid = (int)ceil((float)(WIDTH*HEIGHT) / (float)(threadsPerBlock));

__device__ double d_map(double x, double in_min, double in_max, double out_min, double out_max)
{
	return (x - in_min) * (out_max - out_min) / (in_max - in_min) + out_min;
}

__global__ void cudaFractalGenerate(int width, int height, int maxIterations, double min_valx, double max_valx, double min_valy, double max_valy, float* d_data) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < width*height) {
		int x = idx % width;
		int y = (int)floorf(idx / width);

		double a = d_map(x, 0, width, min_valx, max_valx);
		double b = d_map(y, 0, height, min_valy, max_valy);

		double ca = a;
		double cb = b;
		double n = 0;
		while (n < maxIterations) {
			double aa = a * a - b * b;
			double bb = 2 * a * b;

			a = aa + ca;
			b = bb + cb;
			if (a*a + b * b > 4) {
				break;
			}
			n++;
		}
		double col = d_map(n, 0, maxIterations, 0, 1);
		col = d_map((double)sqrtf(col), 0, 1, 0, 360);
		d_data[idx * 3 + 0] = (float)col;
		d_data[idx * 3 + 1] = 1.0f;
		d_data[idx * 3 + 2] = n == maxIterations ? 0 : 1.0f;
	}
}

double map(double x, double in_min, double in_max, double out_min, double out_max)
{
	return (x - in_min) * (out_max - out_min) / (in_max - in_min) + out_min;
}

void render() {
	hipMalloc((void**)&d_data, sizeof(float) * (WIDTH * HEIGHT * 3));

	cudaFractalGenerate << <blocksPerGrid, threadsPerBlock >> > (WIDTH, HEIGHT, maxIterations, min_valx, max_valx, min_valy, max_valy, d_data);
	hipDeviceSynchronize();

	hipMemcpy(texData, d_data, sizeof(float)*(WIDTH*HEIGHT * 3), hipMemcpyDeviceToHost);
	hipFree(d_data);
}

int main() {
	Display display(WIDTH, HEIGHT, "Fractal Generator");

	Shader shader("./res/");
	shader.Bind();

	float verticies[] = {
		-1.0f, -1.0f,	0.0f, 1.0f,
		 1.0f, -1.0f,	1.0f, 1.0f,
		 1.0f,  1.0f,	1.0f, 0.0f,
		-1.0f,  1.0f,   0.0f, 0.0f
	};

	unsigned int indicies[]{
		0, 1, 2,
		2, 3, 0
	};

	texData = (float*)malloc(sizeof(float)*(WIDTH*HEIGHT * 3));

	unsigned int buffer;
	glGenBuffers(1, &buffer);
	glBindBuffer(GL_ARRAY_BUFFER, buffer);
	glBufferData(GL_ARRAY_BUFFER, sizeof(verticies), verticies, GL_STATIC_DRAW);

	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, sizeof(float) * 4, (void*)0);

	glEnableVertexAttribArray(1);
	glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, sizeof(float) * 4, (void*)8);

	unsigned int i_buffer;
	glGenBuffers(1, &i_buffer);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, i_buffer);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indicies), indicies, GL_STATIC_DRAW);

	unsigned int texture;
	glGenTextures(1, &texture);
	glBindTexture(GL_TEXTURE_2D, texture);

	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_MIRRORED_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_MIRRORED_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	while (!display.isClosed()) {

		if (display.needToDraw) {
			if (maxIterations < 0) {
				maxIterations = 0;
			}
			maxIterations += (((int)maxIterations / 10)+1)*(display.getWheel());
			printf("%f \n", maxIterations);
			render();

			for (int i = 0; i < WIDTH*HEIGHT; i++) {
				hsv color1;
				color1.h = texData[i * 3 + 0];
				color1.s = texData[i * 3 + 1];
				color1.v = texData[i * 3 + 2];
				rgb color2;
				color2 = hsv2rgb(color1);
				texData[i * 3 + 0] = (float)color2.r;
				texData[i * 3 + 1] = (float)color2.g;
				texData[i * 3 + 2] = (float)color2.b;
			}

			glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB8, WIDTH, HEIGHT, 0, GL_RGB, GL_FLOAT, texData);
			display.needToDraw = false;
		}

		glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
		display.Update();
	}

	return 0;
}