// CUDA
#include "hip/hip_runtime.h"
#include ""

// OPEN_GL
#include <OPEN_GL/glew.h>

// SDL2
#include <SDL2/SDL.h>

#include "SDL_Display.h"
#include "shader.h"
#include <stdio.h>

int WIDTH = 500;
int HEIGHT = 500;

float *texData;
int main() {
	Display display(WIDTH, HEIGHT, "Fractal Generator");

	Shader shader("./res/");
	shader.Bind();

	float verticies[] = {
		-1.0f, -1.0f,	0.0f, 1.0f,
		 1.0f, -1.0f,	1.0f, 1.0f,
		 1.0f,  1.0f,	1.0f, 0.0f,
		-1.0f,  1.0f,   0.0f, 0.0f
	};

	unsigned int indicies[] {
		0, 1, 2,
		2, 3, 0
	};

	texData = (float*)malloc(sizeof(float)*(WIDTH*HEIGHT * 3));

	for (int i = 0; i < WIDTH*HEIGHT; i++) {
		texData[i * 3] = 0.0f;
		texData[i * 3 + 1] = 1.0f;
		texData[i * 3 + 2] = 0.7f;
	}

	for (int i = 0; i < 1000; i++) {
		texData[i * 3] = 1.0f;
		texData[i * 3 + 1] = 0.0f;
		texData[i * 3 + 2] = 0.0f;
	}

	unsigned int buffer;
	glGenBuffers(1, &buffer);
	glBindBuffer(GL_ARRAY_BUFFER, buffer);
	glBufferData(GL_ARRAY_BUFFER, sizeof(verticies), verticies, GL_STATIC_DRAW);

	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, sizeof(float) * 4, (void*)0);

	glEnableVertexAttribArray(1);
	glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, sizeof(float) * 4, (void*)8);

	unsigned int i_buffer;
	glGenBuffers(1, &i_buffer);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, i_buffer);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(indicies), indicies, GL_STATIC_DRAW);

	unsigned int texture;
	glGenTextures(1, &texture);
	glBindTexture(GL_TEXTURE_2D, texture);

	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_MIRRORED_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_MIRRORED_REPEAT);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);

	glTexImage2D(GL_TEXTURE_2D, 0, GL_RGB8, WIDTH, HEIGHT, 0, GL_RGB, GL_FLOAT, texData);

	
	while (!display.isClosed()) {
		//glDrawArrays(GL_TRIANGLES, 0, 3);
		glDrawElements(GL_TRIANGLES, 6, GL_UNSIGNED_INT, 0);
		display.Update();
	}

	return 0;
}